#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <iostream>

// CUDA runtime
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>

// private
#include "gpu_main.h"
#include "gpu_func.cuh"




using namespace std;





// Allocates a matrix with random float entries.
void randomInit(float * data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}




/**
 * Run a simple test of matrix multiplication using CUDA
 */
int matrixMultiply(int block_size, dim3 &dimsA, dim3 &dimsB)
{
	// Allocate host memory for matrices A and B
	unsigned int size_A = dimsA.x * dimsA.y;
	unsigned int mem_size_A = sizeof(float) * size_A;
	float * h_A = (float *)malloc(mem_size_A);
	unsigned int size_B = dimsB.x * dimsB.y;
	unsigned int mem_size_B = sizeof(float) * size_B;
	float *h_B = (float *)malloc(mem_size_B);


	// Initialize host memory
	/* initialize random seed: */
	srand (time(NULL));
	randomInit(h_A, size_A);
	randomInit(h_B, size_B);


	// Allocate device memory
	float *d_A, *d_B, *d_C;


	// Allocate host matrix C
	dim3 dimsC(dimsB.x, dimsA.y, 1);
	unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
	float * h_C = (float *) malloc(mem_size_C);


	// Allocate device memory
	hipMalloc((void **) &d_A, mem_size_A);
	hipMalloc((void **) &d_B, mem_size_B);
	hipMalloc((void **) &d_C, mem_size_C);


	// copy host memory to device
	hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice);


	// Setup execution parameters
	dim3 threads(block_size, block_size);
	dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);


	// Create and start timer
	printf("Computing result using CUDA Kernel...\n");

	// Performs warmup operation using matrixMul CUDA kernel
	if (block_size == 16)
	{
		matrixMulCUDA<16><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
	}
	else
	{
		matrixMulCUDA<32><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
	}

	printf("warmup done...\n");

	hipDeviceSynchronize();



	// Allocate CUDA events that we'll use for timing
	hipEvent_t start;
	hipEventCreate(&start);
	hipEvent_t stop;
	hipEventCreate(&stop);
	// Record the start event
	hipEventRecord(start, NULL);

	// Execute the kernel
	int nIter = 300;
	for (int j = 0; j < nIter; j++)
	{
		if (block_size == 16)
		{
			matrixMulCUDA<16><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
		}
		else
		{
			matrixMulCUDA<32><<< grid, threads >>>(d_C, d_A, d_B, dimsA.x, dimsB.x);
		}
	}

	// Record the stop event
	hipEventRecord(stop, NULL);
	// Wait for the stop event to complete
	hipEventSynchronize(stop);
	float msecTotal = 0.0f;
	hipEventElapsedTime(&msecTotal, start, stop);


	// Compute and print the performance
	float msecPerMatrixMul = msecTotal / nIter;
	double flopsPerMatrixMul = 2.0 * (double)dimsA.x * (double)dimsA.y * (double)dimsB.x;
	double gigaFlops = (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
	printf(
		"Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops, WorkgroupSize= %u threads/block\n",
		gigaFlops,
		msecPerMatrixMul,
		flopsPerMatrixMul,
		threads.x * threads.y);



	// Copy result from device to host
	hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);


	// result check
	printf("Checking computed result for correctness: ");
	bool correct = true;

	// test relative error by the formula
	//     |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
	double eps = 1.e-6 ; // machine zero

	// calculate the reference from CPU
	float * ref_C = (float *)malloc(mem_size_C);
	for(int i=0; i<dimsA.y; i++)
	{
		for(int j=0; j<dimsB.x; j++)
		{
			// cal ref_C[i * dimsB.x + j]
			float value = 0;
			for(int k=0; k<dimsA.x; k++)
			{
				value += h_A[dimsA.x * i + k] * h_B[dimsB.x * k + j];
			}
			ref_C[i * dimsB.x + j] = value;
		}
	}

	// compare
	for (int i = 0; i < (int)(dimsC.x * dimsC.y); i++)
	{
		double abs_err = fabs(h_C[i] - ref_C[i]);
		double dot_length = dimsA.x;
		double abs_val = fabs(h_C[i]);
		double rel_err = abs_err/abs_val/dot_length;

		if (rel_err > eps)
		{
			printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i, h_C[i], ref_C[i], eps);
			correct = false;
		}
	}

	printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");


	// Clean up memory
	free(h_A);
	free(h_B);
	free(h_C);
	free(ref_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	printf("\nNOTE: The CUDA Samples are not meant for performance measurements. Results may vary when GPU Boost is enabled.\n");

	// hipDeviceReset causes the driver to clean up all state. While
	// not mandatory in normal operation, it is good practice.  It is also
	// needed to ensure correct operation when the application is being
	// profiled. Calling hipDeviceReset causes all profile data to be
	// flushed before the application exits
	hipDeviceReset();

	if (correct)
	{
		return EXIT_SUCCESS;
	}
	else
	{
		return EXIT_FAILURE;
	}
}



/**
 * GPU Program main
 */
void gpu_main()
{
	printf("[Matrix Multiply Using CUDA] - Starting...\n");

	int block_size = 32;										// 1024 (32*32) is the current block size limit

	dim3 dimsA(5*2*block_size, 5*2*block_size, 1);				// TODO: change scales of input
	dim3 dimsB(5*4*block_size, 5*2*block_size, 1);				// TODO: change scales of input
	printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x, dimsB.y);

	int matrix_result = matrixMultiply(block_size, dimsA, dimsB);

	cout << "MM func result: ";
	cout << matrix_result << endl;

	return;
}


